#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------
This source distribution is placed in the public domain by its author,
Jason Papadopoulos. You may use it for any purpose, free of charge,
without having to notify anyone. I disclaim any responsibility for any
errors.

Optionally, please be nice and tell me if you find this source to be
useful. Again optionally, if you add to the functionality present here
please consider making those additions public too, so that others may 
benefit from your work.	

$Id$
--------------------------------------------------------------------*/

#include "stage1_core_deg6_96.h"

#ifdef __cplusplus
extern "C" {
#endif

__constant__ uint64 pbatch[P_ARRAY_WORDS];

__constant__ uint96 two = {{2, 0, 0}};

/*------------------------------------------------------------------------*/
__device__ p_packed_t *
p_packed_next(p_packed_t *curr)
{
	return (p_packed_t *)((uint64 *)curr + 
			P_PACKED_HEADER_WORDS + 
			3 * (curr->num_roots / 2));
}

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel(q_soa_t *qbatch, 
             uint32 num_q,
	     uint32 num_qroots,
	     uint32 num_p,
	     found_t *found_array)
{
	uint32 my_threadid;
	uint32 num_threads;
	uint32 i, j, k, m;

	my_threadid = blockIdx.x * blockDim.x + threadIdx.x;
	num_threads = gridDim.x * blockDim.x;
	found_array[my_threadid].p = 0;

	for (i = my_threadid; i < num_q; i += num_threads) {
		uint64 q = qbatch->p[i];
		uint96 q2 = wide_sqr48(q);
		uint32 q2_w = montmul32_w(q2.w[0]);
		uint96 q2_r = montmul96_r(q2, q2_w);
		p_packed_t *curr_p = (p_packed_t *)pbatch;
		
		for (j = 0; j < num_p; j++) {
			uint64 p = curr_p->p;
			uint96 p2 = wide_sqr48(p);
			uint64 pinvmodq = modinv64(p, q);

			uint32 num_proots = curr_p->num_roots;
			uint64 lattice_size = curr_p->lattice_size;
			uint96 pinv, tmp;
			uint96 test1;

			test1.w[0] = (uint32)lattice_size;
			test1.w[1] = (uint32)(lattice_size >> 32);
			test1.w[2] = 0;

			tmp = wide_sqr48(pinvmodq);
			tmp = montmul96(tmp, q2_r, q2, q2_w);
			pinv = montmul96(p2, tmp, q2, q2_w);
			pinv = modsub96(two, pinv, q2);
			pinv = montmul96(pinv, tmp, q2, q2_w);
			pinv = montmul96(pinv, q2_r, q2, q2_w);

			for (k = 0; k < 3 * num_qroots; k += 3) {

				uint96 qroot;

				qroot.w[0] = qbatch->roots[k][i];
				qroot.w[1] = qbatch->roots[k+1][i];
				qroot.w[2] = qbatch->roots[k+2][i];

				for (m = 0; m < num_proots; m++) {

					uint96 proot = curr_p->roots[m];
					uint96 res = montmul96(pinv, 
							modsub96(qroot, 
								proot, q2),
							q2, q2_w);

					if (cmp96(res, test1) <= 0) {
						found_t *f = found_array +
								my_threadid;
						f->p = p;
						f->q = q;
						f->offset = res;
						f->proot = proot;
					}
				}
			}

			curr_p = p_packed_next(curr_p);
		}
	}
}

#ifdef __cplusplus
}
#endif
